#include "hip/hip_runtime.h"
#include "flip.cuh"

__global__
void createStates(int N, unsigned int seed, hiprandState_t* states)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x; //One thread per state
    //Don't need to use the lookups here b.c. it doesn't matter if we create too many states
    if (threadID < N)
    {
        hiprand_init(seed, threadID, 0, &states[threadID]);
    }
}

__global__
void wipeArrays(int N, int* qubits, int* syndrome)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x; //One thread per array element
    //Don't need lookups here either
    if (threadID < N)
    {
        qubits[threadID] = 0;
        syndrome[threadID] = 0;
    } 
}

//This works for qubit or syndrome errors
//errorTarget is either qubits or syndrome
//errorProb is p or q
__global__
void applyErrors(int* lookup, hiprandState_t* states, int* errorTarget, float errorProb)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x; //One thread per errorTarget element
    if (lookup[threadID] == 1)
    {
        if (hiprand_uniform(&states[threadID]) < errorProb)
        {
            errorTarget[threadID] = (errorTarget[threadID] + 1) % 2;
        }
    }
}

//Regular deterministic flip
__global__
void flip(int* lookup , int* qubits, int* syndrome, int* faceToEdges)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x; //One thread per qubit
    if (lookup[threadID] == 1)
    {
        int* edges = faceToEdges[threadID];
        int n = 0;
        for (int i=0; i<4; i++)
        {
            if (d_syndrome[edges[i]] == 1) n++;
        }
        if (n > 2) d_qubits[threadID] = (d_qubits[threadID] + 1) % 2;
    }
}

//Probabilistic flip
__global__
void pflip(int* lookup, int* qubits, int* syndrome, int* faceToEdges, hiprandState_t* states)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x; //One thread per qubit
    if (lookup[threadID] == 1)
    {
        int* edges = faceToEdges[threadID];
        int n = 0;
        for (int i=0; i<4; i++)
        {
            if (syndrome[edges[i]] == 1) n++;
        }
        if (n > 2) qubits[threadID] = (qubits[threadID] + 1) % 2;
        if (n == 2) 
        {
            if (hiprand_uniform(&states[threadID]) < 0.5)
            {
                qubits[threadID] = (qubits[threadID] + 1) % 2;
            }
        }
    }
}

__global__
void updateSyndrome(int* lookup , int* qubits, int* syndrome, int* edgeToFaces)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x; //One thread per stabiliser
    if (lookup[threadID] == 1)
    {
        int* faces = edgeToFaces[threadID];
        int parity = 0;
        for (int i=0; i<4; i++)
        {
            if (qubits[faces[i]] == 1) parity = (parity + 1) % 2;
        }
        syndrome[threadID] = parity;
    }
}

__global__
void measureLogicals(int* lookup, int* qubits, int* nOdd, int L, char bounds)
{
    //Just check the reps that run in the Z direction
    //Code and error model are symmetric along all axis
    //so expect performance for other two logical qubits is the same
    int threadID = blockIdx.x * blockDim.x + threadIdx.x; //One thread per logical op rep
    if (lookup[threadID] == 1)
    {
        int qubit = threadID;
        int parity = qubits[qubit];
        //Don't need to check bounds is 'o' or 'c' only here because it was checked earlier
        if (bounds == 'o')
        {
            for (int i=0; i<L-3; ++i)
            {
                qubit += 3*L*L;
                parity = (parity + qubits[qubit]) % 2;
            }
        }
        else
        {
            for (int i=0; i<L; ++i)
            { 
                qubit += 3*L*L;
                parity = (parity + qubits[qubit]) % 2;
            }
        }
        atomicAdd(nOdd, parity);
    }
}
