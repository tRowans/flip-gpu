#include "hip/hip_runtime.h"
#include<random>
#include<iostream>
#include "code.h"
#include "decode.cuh"

int main(int argc, char *argv[])
{
    if (argc != 13)
    {
        std::cout << "Invalid number of arguments." << '\n';
        return 1;
    }

    double pLower = std::atof(argv[1]);  //lower value for error probability p
    double pUpper = std::atof(argv[2]);  //upper value for p
    int nps = std::atoi(argv[3]);        //number of values for p in range pLower <= p <= pUpper
    double beta = std::atof(argv[4]);    //measurement error probability q = beta*p
    int runs = std::atoi(argv[5]);       //number of repeats of simulation
    int cycles = std::atoi(argv[6]);     //code cycles per simulation
    int useBP = std::atoi(argv[7]);      //use BP in decoding? (0 = no, 1 = tanh BP, 2 = min sum BP)
    int bpIters = std::atoi(argv[8]);    //BP iterations per code cycle
    int alpha = std::atof(argv[9]);      //alpha parameter for minsum BP
    int useFlip = std::atoi(argv[10]);    //use flip in decoding? (0 = np, 1 = yes)
    int flipIters = std::atoi(argv[11]); //flip iterations per code cycle
    int pfreq = std::atoi(argv[12]);     //apply p-flip instead of flip every pfreq applications

    double pRange = pUpper - pLower;
    double pStep;
    if (nps == 1) pStep = 0;
    else pStep = pRange/(nps-1); 
    double ps[nps];
    double qs[nps];
    for (int i=0; i<nps; ++i)
    {
        ps[i] = pLower + i*pStep;
        qs[i] = beta*ps[i];
    }
      
    //build code info 
    Code code("parity_check_matrices/lifted_product_[[416,18,20]]",416);

    //for copying out later
    int variablesX[code.N_X] = {};     
    int variablesZ[code.N_Z] = {};
    int factorsX[code.M_X] = {};
    int factorsZ[code.M_Z] = {};
    //pointers for arrays on device
    int *d_variablesX, *d_variablesZ, *d_factorsX, *d_factorsZ;
    int *d_variableDegreesX, *d_variableToFactorsX, *d_variableDegreesZ, *d_variableToFactorsZ;
    int *d_factorDegreesX, *d_factorToVariablesX, *d_factorDegreesZ, *d_factorToVariablesZ;
    int *d_variableToPosX, *d_variableToPosZ, *d_factorToPosX, *d_factorToPosZ;
    //BP message array pointers
    double *d_variableMessagesX, *d_variableMessagesZ, *d_factorMessagesX, *d_factorMessagesZ; 
    double *d_marginalsX, *d_marginalsZ;

    //don't need to copy for these, just set to all zeros on device (later)
    hipMalloc(&d_variablesX, code.N_X*sizeof(int));
    hipMalloc(&d_variablesZ, code.N_Z*sizeof(int));
    hipMalloc(&d_factorsX, code.M_X*sizeof(int));
    hipMalloc(&d_factorsZ, code.M_Z*sizeof(int));

    //these get copied to device from initialised versions in code object
    hipMalloc(&d_variableDegreesX, code.N_X*sizeof(int));
    hipMemcpy(d_variableDegreesX, code.variableDegreesX,
                code.N_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableToFactorsX, code.maxVariableDegreeX*code.N_X*sizeof(int));
    hipMemcpy(d_variableToFactorsX, code.variableToFactorsX[0], 
                code.maxVariableDegreeX*code.N_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableDegreesZ, code.N_Z*sizeof(int));
    hipMemcpy(d_variableDegreesZ, code.variableDegreesZ,
                code.N_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableToFactorsZ, code.maxVariableDegreeZ*code.N_Z*sizeof(int));
    hipMemcpy(d_variableToFactorsZ, code.variableToFactorsZ[0], 
                code.maxVariableDegreeZ*code.N_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorDegreesX, code.M_X*sizeof(int));
    hipMemcpy(d_factorDegreesX, code.factorDegreesX,
                code.M_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorToVariablesX, code.maxFactorDegreeX*code.M_X*sizeof(int));
    hipMemcpy(d_factorToVariablesX, code.factorToVariablesX[0], 
                code.maxFactorDegreeX*code.M_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorDegreesZ, code.M_Z*sizeof(int));
    hipMemcpy(d_factorDegreesZ, code.factorDegreesZ,
                code.M_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorToVariablesZ, code.maxFactorDegreeZ*code.M_Z*sizeof(int));
    hipMemcpy(d_factorToVariablesZ, code.factorToVariablesZ[0], 
                code.maxFactorDegreeZ*code.M_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableToPosX, code.maxVariableDegreeX*code.N_X*sizeof(int));
    hipMemcpy(d_variableToPosX, code.variableToPosX[0],
                code.maxVariableDegreeX*code.N_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableToPosZ, code.maxVariableDegreeZ*code.N_Z*sizeof(int));
    hipMemcpy(d_variableToPosZ, code.variableToPosZ[0],
                code.maxVariableDegreeZ*code.N_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorToPosX, code.maxFactorDegreeX*code.M_X*sizeof(int));
    hipMemcpy(d_factorToPosX, code.factorToPosX[0],
                code.maxFactorDegreeX*code.M_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorToPosZ, code.maxFactorDegreeZ*code.M_Z*sizeof(int));
    hipMemcpy(d_factorToPosZ, code.factorToPosZ[0],
            code.maxFactorDegreeZ*code.M_Z*sizeof(int), hipMemcpyHostToDevice);

    //These also get initialised on device
    hipMalloc(&d_variableMessagesX, code.maxFactorDegreeZ*code.M_Z*sizeof(double));
    hipMalloc(&d_variableMessagesZ, code.maxFactorDegreeX*code.M_X*sizeof(double));
    hipMalloc(&d_factorMessagesX, code.maxVariableDegreeZ*code.N_Z*sizeof(double));
    hipMalloc(&d_factorMessagesZ, code.maxVariableDegreeX*code.N_X*sizeof(double));
    hipMalloc(&d_marginalsX, code.N_X*sizeof(double));
    hipMalloc(&d_marginalsZ, code.N_Z*sizeof(double));

    //setup state array for device-side random number generation
    std::random_device rd{};
    hiprandState_t *d_states;
    //just need at least as many states as the largest array
    if (code.N_X > code.N_Z) 
    {
        hipMalloc(&d_states, code.N_X*sizeof(hiprandState_t));
        createStates<<<(code.N_X+255)/256,256>>>(code.N_X, rd(), d_states);
    }
    else 
    {
        hipMalloc(&d_states, code.N_Z*sizeof(hiprandState_t));
        createStates<<<(code.N_Z+255)/256,256>>>(code.N_Z, rd(), d_states);
    }
    hipDeviceSynchronize();

    for (int i=0; i<nps; ++i)
    {
        double llrp0 = log10((1-ps[i])/ps[i]);
        double llrq0 = log10((1-qs[i])/qs[i]);
        for (int run=0; run<runs; ++run)
        {
            //set qubits to all zeros 
            wipeArray<<<(code.N_X+255)/256,256>>>(code.N_X, d_variablesX);
            wipeArray<<<(code.N_Z+255)/256,266>>>(code.N_Z, d_variablesZ);
            hipDeviceSynchronize();
        
            for (int cycle=0; cycle<cycles; ++cycle) 
            {
                depolErrors<<<(code.nQubits+255)/256,256>>>(code.nQubits, d_states, d_variablesX, d_variablesZ, ps[i]);                 //qubit errors
                measErrors<<<(code.N_X+255)/256,256>>>(code.nQubits, code.nChecksZ, d_states, d_variablesX, qs[i]);                     //Z measurement errors
                measErrors<<<(code.N_Z+255)/256,256>>>(code.nQubits, code.nChecksX, d_states, d_variablesZ, qs[i]);                     //X measurement errors
                hipDeviceSynchronize();
                calculateSyndrome<<<(code.M_Z+255)/256,256>>>(code.M_Z, d_variablesX, d_factorsZ,                                       //calculate checks
                                        d_factorToVariablesZ, d_factorDegreesZ, code.maxFactorDegreeZ);                                 //(inc. metachecks)
                calculateSyndrome<<<(code.M_X+255)/256,256>>>(code.M_X, d_variablesZ, d_factorsX, 
                                        d_factorToVariablesX, d_factorDegreesX, code.maxFactorDegreeX);
                hipDeviceSynchronize();

                //prepare initial distributions for BP
                initVariableMessages<<<(code.M_Z+255)/256,256>>>(code.M_Z, code.nChecksZ, d_variableMessagesX, d_factorDegreesZ,
                        code.maxFactorDegreeZ, llrp0, llrq0);
                initVariableMessages<<<(code.M_X+255)/256,256>>>(code.M_X, code.nChecksX, d_variableMessagesZ, d_factorDegreesX,
                        code.maxFactorDegreeX, llrp0, llrq0);
                hipDeviceSynchronize();

                //BP
                if (useBP)
                {
                    for (int iter=0; iter<bpIters; ++iter)
                    {
                        if (useBP == 1)
                        {
                            updateFactorMessagesTanh<<<(code.M_Z+255)/256,256>>>(code.M_Z, d_variableMessagesX, d_factorMessagesZ, d_factorsZ,
                                    d_factorToVariablesZ, d_factorDegreesZ, code.maxFactorDegreeZ, d_factorToPosZ, code.maxVariableDegreeX);
                            updateFactorMessagesTanh<<<(code.M_X+255)/256,256>>>(code.M_X, d_variableMessagesZ, d_factorMessagesX, d_factorsX,
                                    d_factorToVariablesX, d_factorDegreesX, code.maxFactorDegreeX, d_factorToPosX, code.maxVariableDegreeZ);
                        }
                        else if (useBP == 2)
                        {
                            updateFactorMessagesMinSum<<<(code.M_Z+255)/256,256>>>(alpha, code.M_Z, d_variableMessagesX, d_factorMessagesZ, d_factorsZ,
                                    d_factorToVariablesZ, d_factorDegreesZ, code.maxFactorDegreeZ, d_factorToPosZ, code.maxVariableDegreeX);
                            updateFactorMessagesMinSum<<<(code.M_X+255)/256,256>>>(alpha, code.M_X, d_variableMessagesZ, d_factorMessagesX, d_factorsX,
                                    d_factorToVariablesX, d_factorDegreesX, code.maxFactorDegreeX, d_factorToPosX, code.maxVariableDegreeZ);
                        }
                        hipDeviceSynchronize();
                        updateVariableMessages<<<(code.N_X+255)/256,256>>>(code.N_X, code.nQubits, d_factorMessagesZ, d_variableMessagesX, 
                                d_variableToFactorsX, d_variableDegreesX, code.maxVariableDegreeX, d_variableToPosX, code.maxFactorDegreeZ, llrp0, llrq0);
                        updateVariableMessages<<<(code.N_Z+255)/256,256>>>(code.N_Z, code.nQubits, d_factorMessagesX, d_variableMessagesZ,
                                d_variableToFactorsZ, d_variableDegreesZ, code.maxVariableDegreeZ, d_variableToPosZ, code.maxFactorDegreeX, llrp0, llrq0);
                        hipDeviceSynchronize();
                    }
                    calcMarginals<<<(code.N_X+255)/256,256>>>(code.N_X, code.nQubits, d_marginalsX, d_factorMessagesZ, 
                                                                d_variableDegreesX, code.maxVariableDegreeX, llrp0, llrq0);
                    calcMarginals<<<(code.N_Z+255)/256,256>>>(code.N_Z, code.nQubits, d_marginalsZ, d_factorMessagesX, 
                                                                d_variableDegreesZ, code.maxVariableDegreeZ, llrp0, llrq0);
                    hipDeviceSynchronize();    
                    bpCorrection<<<(code.N_X+255)/256,256>>>(code.nQubits, code.nChecksZ, d_marginalsX, d_variablesX, d_factorsZ,
                            d_variableToFactorsX, d_variableDegreesX, code.maxVariableDegreeX);
                    bpCorrection<<<(code.N_Z+255)/256,256>>>(code.nQubits, code.nChecksX, d_marginalsZ, d_variablesZ, d_factorsX,
                            d_variableToFactorsZ, d_variableDegreesZ, code.maxVariableDegreeZ);
                    hipDeviceSynchronize();
                }
                //flip
                if (useFlip)
                {
                    for (int iter=0; iter<flipIters; ++iter)
                    {
                        //if we used BP we can run pflip straight away, otherwise do some normal flip first
                        if ((useBP == 0 && (iter+1) % pfreq == 0) || (useBP == 1 && iter % pfreq == 0))
                        {
                            pflip<<<(code.N_X+255)/256,256>>>(code.nQubits, d_states, d_variablesX, d_factorsZ,
                                        d_variableToFactorsX, d_variableDegreesX, code.maxVariableDegreeX);
                            pflip<<<(code.N_Z+255)/256,256>>>(code.nQubits, d_states, d_variablesZ, d_factorsX,
                                        d_variableToFactorsZ, d_variableDegreesZ, code.maxVariableDegreeZ);
                        }
                        else
                        {
                            flip<<<(code.N_X+255)/256,256>>>(code.nQubits, d_variablesX, d_factorsZ, 
                                    d_variableToFactorsX, d_variableDegreesX, code.maxVariableDegreeX);
                            flip<<<(code.N_Z+255)/256,256>>>(code.nQubits, d_variablesZ, d_factorsX,
                                    d_variableToFactorsZ, d_variableDegreesZ, code.maxVariableDegreeZ);
                        }
                        hipDeviceSynchronize();
                    }
                }
            }

            hipMemcpy(variablesX, d_variablesX, code.N_X*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(factorsZ, d_factorsZ, code.M_Z*sizeof(int), hipMemcpyDeviceToHost);
            std::cout << ps[i] << ',' << run << ",q,X";
            for (int j=0; j<code.nQubits; ++j) std::cout << ',' << variablesX[j];
            std::cout << '\n';
            std::cout << ps[i] << ',' << run << ",s,Z";
            for (int j=0; j<code.nChecksZ; ++j) std::cout << ',' << factorsZ[j];
            std::cout << '\n';
            hipMemcpy(variablesZ, d_variablesZ, code.N_Z*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(factorsX, d_factorsX, code.M_X*sizeof(int), hipMemcpyDeviceToHost);
            std::cout << ps[i] << ',' << run << ",q,Z";
            for (int j=0; j<code.nQubits; ++j) std::cout << ',' << variablesZ[j];
            std::cout << '\n';
            std::cout << ps[i] << ',' << run << ",s,X";
            for (int j=0; j<code.nChecksX; ++j) std::cout << ',' << factorsX[j];
            std::cout << '\n';
        }
    }

    hipFree(d_variablesX);
    hipFree(d_variablesZ);
    hipFree(d_factorsX);
    hipFree(d_factorsZ);
    hipFree(d_variableDegreesX);
    hipFree(d_variableToFactorsX);
    hipFree(d_variableDegreesZ);
    hipFree(d_variableToFactorsZ);
    hipFree(d_factorDegreesX);
    hipFree(d_factorToVariablesX);
    hipFree(d_factorDegreesZ);
    hipFree(d_factorToVariablesZ);
    hipFree(d_variableToPosX);
    hipFree(d_variableToPosZ);
    hipFree(d_factorToPosX);
    hipFree(d_factorToPosZ);
    hipFree(d_variableMessagesX);
    hipFree(d_variableMessagesZ);
    hipFree(d_factorMessagesX);
    hipFree(d_factorMessagesZ);
    hipFree(d_marginalsX);
    hipFree(d_marginalsZ);

    return 0;
}
