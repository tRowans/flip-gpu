#include "hip/hip_runtime.h"
#include<random>
#include<iostream>
#include "code.h"
#include "decode.cuh"

int main(int argc, char *argv[])
{
    if (argc != 13)
    {
        std::cout << "Invalid number of arguments." << '\n';
        return 1;
    }

    double pLower = std::atof(argv[1]);  //lower value for error probability p
    double pUpper = std::atof(argv[2]);  //upper value for p
    int nps = std::atoi(argv[3]);        //number of values for p in range pLower <= p <= pUpper
    double beta = std::atof(argv[4]);    //measurement error probability q = beta*p
    int runs = std::atoi(argv[5]);       //number of repeats of simulation
    int cycles = std::atoi(argv[6]);     //code cycles per simulation
    int useBP = std::atoi(argv[7]);      //use BP in decoding? (0 = no, 1 = tanh BP, 2 = min sum BP)
    int bpIters = std::atoi(argv[8]);    //BP iterations per code cycle
    int alpha = std::atof(argv[9]);      //alpha parameter for minsum BP
    int useFlip = std::atoi(argv[10]);    //use flip in decoding? (0 = np, 1 = yes)
    int flipIters = std::atoi(argv[11]); //flip iterations per code cycle
    int pfreq = std::atoi(argv[12]);     //apply p-flip instead of flip every pfreq applications

    double pRange = pUpper - pLower;
    double pStep;
    if (nps == 1) pStep = 0;
    else pStep = pRange/(nps-1); 
    double ps[nps];
    double qs[nps];
    for (int i=0; i<nps; ++i)
    {
        ps[i] = pLower + i*pStep;
        qs[i] = beta*ps[i];
    }
      
    //build code info 
    Code code("parity_check_matrices/lifted_product_[[416,18,20]]");

    //for copying out later
    int qubitsX[code.nQubits] = {};     
    int qubitsZ[code.nQubits] = {};
    int syndromeX[code.M_X] = {};       //using M_X/M_Z rather than nChecksX/Z so we get entries for metachecks
    int syndromeZ[code.M_Z] = {};       //which will always be +1. This simplifies some BP functions
    //pointers for arrays on device
    int *d_qubitsX, *d_qubitsZ, *d_syndromeX, *d_syndromeZ;
    int *d_variableDegreesX, *d_variableToFactorsX, *d_variableDegreesZ, *d_variableToFactorsZ;
    int *d_factorDegreesX, *d_factorToVariablesX, *d_factorDegreesZ, *d_factorToVariablesZ;
    int *d_variableToPosX, *d_variableToPosZ, *d_factorToPosX, *d_factorToPosZ;
    //BP message array pointers
    double *d_variableMessagesX, *d_variableMessagesZ, *d_factorMessagesX, *d_factorMessagesZ; 
    double *d_marginalsX, *d_marginalsZ;

    //don't need to copy for these, just set to all zeros on device (later)
    hipMalloc(&d_qubitsX, code.nQubits*sizeof(int));
    hipMalloc(&d_qubitsZ, code.nQubits*sizeof(int));
    hipMalloc(&d_syndromeX, code.M_X*sizeof(int));
    hipMalloc(&d_syndromeZ, code.M_Z*sizeof(int));

    //these get copied to device from initialised versions in code object
    hipMalloc(&d_variableDegreesX, code.N_X*sizeof(int));
    hipMemcpy(d_variableDegreesX, code.variableDegreesX,
                code.N_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableToFactorsX, code.maxVariableDegreeX*code.N_X*sizeof(int));
    hipMemcpy(d_variableToFactorsX, code.variableToFactorsX[0], 
                code.maxVariableDegreeX*code.N_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableDegreesZ, code.N_Z*sizeof(int));
    hipMemcpy(d_variableDegreesZ, code.variableDegreesZ,
                code.N_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableToFactorsZ, code.maxVariableDegreeZ*code.N_Z*sizeof(int));
    hipMemcpy(d_variableToFactorsZ, code.variableToFactorsZ[0], 
                code.maxVariableDegreeZ*code.N_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorDegreesX, code.M_X*sizeof(int));
    hipMemcpy(d_factorDegreesX, code.factorDegreesX,
                code.M_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorToVariablesX, code.maxFactorDegreeX*code.M_X*sizeof(int));
    hipMemcpy(d_factorToVariablesX, code.factorToVariablesX[0], 
                code.maxFactorDegreeX*code.M_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorDegreesZ, code.M_Z*sizeof(int));
    hipMemcpy(d_factorDegreesZ, code.factorDegreesZ,
                code.M_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorToVariablesZ, code.maxFactorDegreeZ*code.M_Z*sizeof(int));
    hipMemcpy(d_factorToVariablesZ, code.factorToVariablesZ[0], 
                code.maxFactorDegreeZ*code.M_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableToPosX, code.maxVariableDegreeX*code.N_X*sizeof(int));
    hipMemcpy(d_variableToPosX, code.variableToPosX[0],
                code.maxVariableDegreeX*code.N_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_variableToPosZ, code.maxVariableDegreeZ*code.N_Z*sizeof(int));
    hipMemcpy(d_variableToPosZ, code.variableToPosZ[0],
                code.maxVariableDegreeZ*code.N_Z*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorToPosX, code.maxFactorDegree*code.M_X*sizeof(int));
    hipMemcpy(d_factorToPosX, code.factorToPosX[0],
                code.maxFactorDegreeX*code.M_X*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_factorToPosZ, code.maxFactorDegree*code.M_Z*sizeof(int));
    hipMemcpy(d_factorToPosZ, code.factorToPosZ[0],
            code.maxFactorDegreeZ*code.M_Z*sizeof(int), hipMemcpyHostToDevice);

    //These also get initialised on device
    hipMalloc(&d_variableMessagesX, code.maxFactorDegreeX*code.M_X*sizeof(double));
    hipMalloc(&d_variableMessagesZ, code.maxFactorDegreeZ*code.M_Z*sizeof(double));
    hipMalloc(&d_factorMessagesX, code.maxVariableDegreeX*code.N_X*sizeof(double));
    hipMalloc(&d_factorMessagesZ, code.maxVariableDegreeZ*code.N_Z*sizeof(double));
    hipMalloc(&d_marginalsX, code.N_X*sizeof(double));
    hipMalloc(&d_marginalsZ, code.N_Z*sizeof(double));

    //setup state array for device-side random number generation
    std::random_device rd{};
    hiprandState_t *d_states;
    //just need at least as many states as the largest array
    if (code.N_X > code.N_Z) 
    {
        hipMalloc(&d_states, code.N_X*sizeof(hiprandState_t));
        createStates<<<(code.N_X+255)/256,256>>>(code.N_X, rd(), d_states);
    }
    else 
    {
        hipMalloc(&d_states, code.N_Z*sizeof(hiprandState_t));
        createStates<<<(code.N_Z+255)/256,256>>>(code.N_Z, rd(), d_states);
    }
    hipDeviceSynchronize();

    for (int i=0; i<nps; ++i)
    {
        double llrp0 = log10((1-ps[i])/ps[i]);
        double llrq0 = log10((1-qs[i])/qs[i]);
        for (int run=0; run<runs; ++run)
        {
            //set qubits to all zeros 
            wipeArray<<<(code.nQubits+255)/256,256>>>(code.nQubits, d_qubitsX);
            wipeArray<<<(code.nQubits+255)/256,266>>>(code.nQubits, d_qubitsZ);
            hipDeviceSynchronize();
        
            for (int cycle=0; cycle<cycles; ++cycle) 
            {
                depolErrors<<<(code.nQubits+255)/256,256>>>(code.nQubits, d_states, d_qubitsX, d_qubitsZ, ps[i]);                       //qubit errors
                hipDeviceSynchronize();
                calculateSyndrome<<<(code.M_Z+255)/256,256>>>(code.M_Z, d_qubitsX, d_syndromeZ, d_zCheckToBits, code.maxCheckDegreeZ);  //measure stabilisers
                calculateSyndrome<<<(code.M_X+255)/256,256>>>(code.M_X, d_qubitsZ, d_syndromeX, d_xCheckToBits, code.maxCheckDegreeX);
                hipDeviceSynchronize();
                arrayErrors<<<(code.M_Z+255)/256,256>>>(code.M_Z, d_states, d_syndromeZ, qs[i]);                                        //measurement errors
                arrayErrors<<<(code.M_X+255)/256,256>>>(code.M_X, d_states, d_syndromeX, qs[i]);
                hipDeviceSynchronize();

                //prepare initial distributions for BP
                initVariableMessages<<<(code.M_Z+255)/256,256>>>(code.M_Z, code.nChecksZ, d_variableMessagesX, d_factorDegreesZ,
                        code.maxFactorDegreeZ, llrp0, llrq0);
                initVariableMessages<<<(code.M_X+255)/256,256>>>(code.M_X, code.nChecksX, d_variableMessagesZ, d_factorDegreesX,
                        code.maxFactorDegreeX, llrp0, llrq0);
                hipDeviceSynchronize();

                //BP
                if (useBP)
                {
                    for (int iter=0; iter<bpIters; ++iter)
                    {
                        if (useBP == 1)
                        {
                            updateFactorMessagesTanh<<<(code.M_Z+255)/256,256>>>(code.M_Z, d_variableMessagesX, d_factorMessagesZ, d_syndromeZ,
                                    d_factorToVariablesZ, d_factorDegreesZ, code.maxFactorDegreeZ, d_factorToPosZ, code.maxVariableDegreeX);
                            updateFactorMessagesTanh<<<(code.M_X+255)/256,256>>>(code.M_X, d_variableMessagesZ, d_factorMessagesX, d_syndromeX,
                                    d_factorToVariablesX, d_factorDegreesX, code.maxFactorDegreeX, d_factorToPosX, code.maxVariableDegreeZ);
                        }
                        else if (useBP == 2)
                        {
                            updateFactorMessagesMinSum<<<(code.M_Z+255)/256,256>>>(alpha, code.M_Z, d_variableMessagesX, d_factorMessagesZ, d_syndromeZ,
                                    d_factorToVariablesZ, d_factorDegreesZ, code.maxFactorDegreeZ, d_factorToPosZ, code.maxVariableDegreeX);
                            updateFactorMessagesMinSum<<<(code.M_X+255)/256,256>>>(alpha, code.M_X, d_variableMessagesZ, d_factorMessagesX, d_syndromeX,
                                    d_factorToVariablesX, d_factorDegreesX, code.maxFactorDegreeX, d_factorToPosX, code.maxVariableDegreeZ);
                        }
                        hipDeviceSynchronize();
                        updateVariableMessages<<<(code.N_X+255)/256,256>>>(code.N_X, code.nQubits, d_factorMessagesZ, d_variableMessagesX, 
                                d_variableToFactorsX, d_variableDegreesX, code.maxVariableDegreeX, d_variableToPosX, code.maxFactorDegreeZ, llrp0, llrq0);
                        updateVariableMessages<<<(code.N_Z+255)/256,256>>>(code.N_Z, code.nQubits, d_factorMessagesX, d_variableMessagesZ,
                                d_variableToFactorsZ, d_variableDegreesZ, code.maxVariableDegreeZ, d_variableToPosZ, code.maxFactorDegreeX, llrp0, llrq0);
                        hipDeviceSynchronize();
                    }
                    calcMarginals<<<(code.N_X+255)/256,256>>>(code.N_X, code.nQubits, d_marginalsX, d_factorMessagesZ, llrp0, llrq0);
                    calcMarginals<<<(code.N_Z+255)/256,256>>>(code.N_Z, code.nQubits, d_marginalsZ, d_factorMessagesX, llrp0, llrq0);
                    hipDeviceSynchronize();    
                    bpCorrection<<<(code.N_X+255)/256,256>>>(code.N_X, code.nQubits, code.nChecksZ, d_marginalsX, d_qubits, d_syndrome,
                            d_variableToFactorsX, d_variableDegreesX, code.maxVariableDegreeX);
                    bpCorrection<<<(code.N_Z+255)/256,256>>>(code.N_Z, code.nQubits, code.nChecksX, d_marginalsZ, d_qubits, d_syndrome,
                            d_variableToFactorsZ, d_variableDegreesZ, code.maxVariableDegreeZ);
                    hipDeviceSynchronize();
                }
                //flip
                if (useFlip)
                {
                    for (int iter=0; iter<flipIters; ++iter)
                    {
                        //if we used BP we can run pflip straight away, otherwise do some normal flip first
                        if ((useBP == 0 && (iter+1) % pfreq == 0) || (useBP == 1 && iter % pfreq == 0))
                        {
                            pflip<<<(code.N_X+255)/256,256>>>(code.nQubits, d_states, d_qubitsX, d_syndromeZ,
                                        d_variableToFactorsX, d_variableDegreesX, code.maxVariableDegreeX);
                            pflip<<<(code.N_Z+255)/256,256>>>(code.nQubits, d_states, d_qubitsZ, d_syndromeX,
                                        d_variableToFactorsZ, d_variableDegreesZ, code.maxVariableDegreeZ);
                        }
                        else
                        {
                            flip<<<(code.N_X+255)/256,256>>>(code.nQubits, d_qubitsX, d_syndromeZ, 
                                    d_variableToFactorsX, d_variableDegreesX, code.maxVariableDegreeX);
                            flip<<<(code.N_Z+255)/256,256>>>(code.nQubits, d_qubitsZ, d_syndromeX,
                                    d_variableToFactorsZ, d_variableDegreesZ, code.maxVariableDegreeZ);
                        }
                        hipDeviceSynchronize();
                    }
                }
            }

            hipMemcpy(qubitsX, d_qubitsX, code.nQubits*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(syndromeZ, d_syndromeZ, code.M_Z*sizeof(int), hipMemcpyDeviceToHost);
            std::cout << ps[i] << ',' << run << ",q,X";
            for (int j=0; j<code.nQubits; ++j) std::cout << ',' << qubitsX[j];
            std::cout << '\n';
            std::cout << ps[i] << ',' << run << ",s,Z";
            for (int j=0; j<code.M_Z; ++j) std::cout << ',' << syndromeZ[j];
            std::cout << '\n';
            hipMemcpy(qubitsZ, d_qubitsZ, code.nQubits*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(syndromeX, d_syndromeX, code.M_X*sizeof(int), hipMemcpyDeviceToHost);
            std::cout << ps[i] << ',' << run << ",q,Z";
            for (int j=0; j<code.nQubits; ++j) std::cout << ',' << qubitsZ[j];
            std::cout << '\n';
            std::cout << ps[i] << ',' << run << ",s,X";
            for (int j=0; j<code.M_X; ++j) std::cout << ',' << syndromeX[j];
            std::cout << '\n';
        }
    }

    hipFree(d_qubitsX);
    hipFree(d_qubitsZ);
    hipFree(d_syndromeX);
    hipFree(d_syndromeZ);
    hipFree(d_variableDegreesX);
    hipFree(d_variableToFactorsX);
    hipFree(d_variableDegreesZ);
    hipFree(d_variableToFactorsZ);
    hipFree(d_factorDegreesX);
    hipFree(d_factorToVariablesX);
    hipFree(d_factorDegreesZ);
    hipFree(d_factorToVariablesZ);
    hipFree(d_variableToPosX);
    hipFree(d_variableToPosZ);
    hipFree(d_factorToPosX);
    hipFree(d_factorToPosZ);
    hipFree(d_variableMessagesX);
    hipFree(d_variableMessagesZ);
    hipFree(d_factorMessagesX);
    hipFree(d_factorMessagesZ);
    hipFree(d_marginalsX);
    hipFree(d_marginalsZ);

    return 0;
}
