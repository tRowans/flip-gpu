#include "decode.cuh"
#include "decode_wrappers.cuh"

//Only used for testing of functions as nvcc doesn't like compiling gtest

void wipeArrayWrap(int N, int* array)
{
    int *d_array;
    hipMalloc(&d_array, N*sizeof(int));
    hipMemcpy(d_array, array, N*sizeof(int), hipMemcpyHostToDevice);
    wipeArray<<<(N+255)/256,256>>>(N, d_array);
    hipMemcpy(array, d_array, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
}

void applyErrorsWrap(int N, unsigned int seed, int* lookup, int* errorTarget, float errorProb)
{
    int *d_lookup, *d_errorTarget;
    hipMalloc(&d_lookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_errorTarget, N*sizeof(int));
    hipMemcpy(d_lookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_errorTarget, errorTarget, N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    applyErrors<<<(N+255)/256,256>>>(d_lookup, d_states, d_errorTarget, errorProb);
    hipMemcpy(errorTarget, d_errorTarget, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_lookup);
    hipFree(d_errorTarget);
}

void calculateSyndromeWrap(int N, int* lookup, int* qubits, int* syndrome, int** edgeToFaces)
{
    int *d_lookup, *d_qubits, *d_syndrome, *d_edgeToFaces;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_lookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    calculateSyndrome<<<(N+255)/256,256>>>(d_lookup, d_qubits, d_syndrome, d_edgeToFaces);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_lookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_edgeToFaces);
}

void flipWrap(int N, int* qLookup, int* sLookup, int* qubits, int* syndrome, int** faceToEdges)
{
    int *d_qLookup, *d_sLookup, *d_qubits, *d_syndrome, *d_faceToEdges;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    flip<<<(N+255)/256,256>>>(d_qLookup, d_sLookup, d_qubits, d_syndrome, d_faceToEdges);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_faceToEdges);
}

void pflipWrap(int N, unsigned int seed, int* qLookup, int* sLookup, int* qubits, int* syndrome, int** faceToEdges)
{
    int *d_qLookup, *d_sLookup, *d_qubits, *d_syndrome, *d_faceToEdges;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    pflip<<<(N+255)/256,256>>>(d_qLookup, d_sLookup, d_qubits, d_syndrome, d_faceToEdges, d_states);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_faceToEdges);
}

void updateSyndromeMessagesWrap(int N, int* lookup, int* qubitMessages, int* syndrome, int* syndromeMessages, int** edgeToFaces, int** faceToEdges)
{
    int *d_sLookup, *d_qubitMessages, *d_syndrome, *d_syndromeMessages, *d_edgeToFaces, *d_faceToEdges;
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_qubitMessages, 8*N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_syndromeMessages, 8*N*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMemcpy(d_sLookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qubitMessages, qubitMessages, 8*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndromeMessages, syndromeMessages, 8*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    updateSyndromeMessages<<<((N+255)/256,256)>>>(d_sLookup, d_qubitMessages, d_syndrome, d_syndromeMessages, d_edgeToFaces, d_faceToEdges);
    hipMemcpy(syndromeMessages, d_syndromeMessages, 8*N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_sLookup);
    hipFree(d_qubitMessages);
    hipFree(d_syndrome);
    hipFree(d_syndromeMessages);
    hipFree(d_edgeToFaces);
    hipFree(d_faceToEdges);
}

void updateQubitMessagesWrap(int N, int* lookup, int* qubitMessages, int* syndromeMessages, int** faceToEdges, int** edgeToFaces, int p)
{
    int *d_qLookup, *d_qubitMessages, *d_syndromeMessages, *d_faceToEdges, *d_edgeToFaces;
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_qubitMessages, 8*N*sizeof(int));
    hipMalloc(&d_syndromeMessages, 8*N*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMemcpy(d_qLookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndromeMessages, syndromeMessages, 8*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgetoFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    updateQubitMessages<<<((N+255)/256,256)>>>(d_qLookup, d_qubitMessages, d_syndromeMesssages, d_faceToEdges, d_edgeToFaces, p);
    hipMemcpy(qubitMessages, d_qubitMessages, 8*N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_qubitMessages);
    hipFree(d_syndromeMessages);
    hipFree(d_faceToEdges);
    hipFree(d_edgeToFaces);
}

void calcMarginalsWrap(int N, int* lookup, int* qubits, int* qubitMarginals, int* syndromeMessages, int p)
{
    int *d_qLookup, *d_qubits, *d_qubitMarginals, *d_syndromeMessages;
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_qubitMarginals, N*sizeof(int));
    hipMalloc(&d_syndromeMessages, 8*N*sizeof(int));
    hipMemcpy(d_qLookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qubitMarginals, qubitMarginals, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndromeMessages, syndromeMessages, 8*N*sizeof(int), hipMemcpyHostToDevice);
    calcMarginals<<<((N+255)/256,256)>>>(d_qLookup, d_qubits, d_qubitMarginals, d_syndromeMessages, p);
    hipMemcpy(qubitMarginals, d_qubitMarginals, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_qubits);
    hipFree(d_qubitsMarginals);
    hipFree(d_syndromeMessages);
}

void bpCorrectionWrap(int N, int* lookup, int* qubits, int* qubitMarginals)
{
    int *d_qLookup, *d_qubits, *d_qubitMarginals;
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_qubitMarginals, N*sizeof(int));
    hipMemcpy(d_qLookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qubitMarginals, qubitMarginals, N*sizeof(int), hipMemcpyHostToDevice);
    bpCorrection<<<((N+255)/256,256)>>>(lookup, qubits, qubitMarginals);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_qubits);
    hipFree(d_qubitMarginals);
}

void measureLogicalsWrap(int N, int* lookup, int* qubits, int &nOdd, int L, char bounds)
{
    int *d_lookup, *d_qubits, *d_nOdd;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_lookup, ((3*L*L+63)/64)*64*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lookup, lookup, ((3*L*L+63)/64)*64*sizeof(int), hipMemcpyHostToDevice);
    hipMallocManaged(&d_nOdd, sizeof(int));
    *d_nOdd = 0;
    measureLogicals<<<(3*L*L+63)/64,64>>>(d_lookup, d_qubits, d_nOdd, L, bounds);
    hipDeviceSynchronize();
    nOdd = *d_nOdd;
    hipFree(d_lookup);
    hipFree(d_qubits);
    hipFree(d_nOdd);
}
