#include "decode.cuh"
#include "decode_wrappers.cuh"

//Only used for testing of functions as nvcc doesn't like compiling gtest

void wipeArrayWrap(int maxIndex, int* array)
{
    int *d_array;
    hipMalloc(&d_array, maxIndex*sizeof(int));
    hipMemcpy(d_array, array, maxIndex*sizeof(int), hipMemcpyHostToDevice);
    wipeArray<<<(maxIndex+255)/256,256>>>(maxIndex, d_array);
    hipMemcpy(array, d_array, maxIndex*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
}

void depolErrorsWrap(int N_X, int N_Z, int nQubits, unsigned int seed, int* variablesX, int* variablesZ, float errorProb)
{
    int *d_variablesX, *d_variablesZ;
    hipMalloc(&d_variablesX, N_X*sizeof(int));
    hipMemcpy(d_variablesX, variablesX, N_X*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&d_variablesZ, N_Z*sizeof(int));
    hipMemcpy(d_variablesZ, variablesZ, N_Z*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    if (N_X > N_Z)
    {
        hipMalloc(&d_states, N_X*sizeof(hiprandState_t));
        createStates<<<(N_X+255)/256,256>>>(N_X, seed, d_states);
    }
    else
    {
        hipMalloc(&d_states, N_Z*sizeof(hiprandState_t));
        createStates<<<(N_Z+255)/256,256>>>(N_X, seed, d_states);
    }
    depolErrors<<<(nQubits+255)/256,256>>>(nQubits, d_states, d_variablesX, d_variablesZ, errorProb);
    hipMemcpy(variablesX, d_variablesX, N_X*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(variablesZ, d_variablesZ, N_Z*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_variablesX);
    hipFree(d_variablesZ);
}

void measErrorsWrap(int nQubits, int nChecks, unsigned int seed, int* variables, float errorProb)
{
    int *d_variables;
    int N = nQubits + nChecks;
    hipMalloc(&d_variables, N*sizeof(int));
    hipMemcpy(d_variables, variables, N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    measErrors<<<(N+255)/256,256>>>(nQubits, nChecks, d_states, d_variables, errorProb);
    hipMemcpy(variables, d_variables, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_variables);
}

void calculateSyndromeWrap(int N, int M, int* variables, int* factors, int** factorToVariables, int* factorDegrees, int maxFactorDegree)
{
    int *d_variables, *d_factors, *d_factorToVariables, *d_factorDegrees;
    hipMalloc(&d_variables, N*sizeof(int));
    hipMalloc(&d_factors, M*sizeof(int));
    hipMalloc(&d_factorToVariables, maxFactorDegree*M*sizeof(int));
    hipMalloc(&d_factorDegrees, M*sizeof(int));
    hipMemcpy(d_variables, variables, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factors, factors, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToVariables, factorToVariables[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorDegrees, factorDegrees, M*sizeof(int), hipMemcpyHostToDevice);
    calculateSyndrome<<<(M+255)/256,256>>>(M, d_variables, d_factors, d_factorToVariables, d_factorDegrees, maxFactorDegree);
    hipMemcpy(variables, d_variables, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(factors, d_factors, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_variables);
    hipFree(d_factors);
    hipFree(d_factorToVariables);
    hipFree(d_factorDegrees);
}

void flipWrap(int N, int M, int nQubits, int nChecks, int* variables, int* factors, int** variableToFactors, int* variableDegrees, int maxVariableDegree)
{
    int *d_variables, *d_factors, *d_variableToFactors, *d_variableDegrees;
    hipMalloc(&d_variables, N*sizeof(int));
    hipMalloc(&d_factors, M*sizeof(int));
    hipMalloc(&d_variableToFactors, maxVariableDegree*N*sizeof(int));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMemcpy(d_variables, variables, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factors, factors, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToFactors, variableToFactors[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    flip<<<(nQubits+255)/256,256>>>(nQubits, d_variables, d_factors, d_variableToFactors, d_variableDegrees, maxVariableDegree);
    hipMemcpy(variables, d_variables, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(factors, d_factors, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_variables);
    hipFree(d_factors);
    hipFree(d_variableToFactors);
    hipFree(d_variableDegrees);
}

void pflipWrap(int N, int M, int nQubits, int nChecks, unsigned int seed, int* variables, int* factors, 
        int** variableToFactors, int* variableDegrees, int maxVariableDegree)
{
    int *d_variables, *d_factors, *d_variableToFactors, *d_variableDegrees;
    hipMalloc(&d_variables, N*sizeof(int));
    hipMalloc(&d_factors, M*sizeof(int));
    hipMalloc(&d_variableToFactors, maxVariableDegree*N*sizeof(int));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMemcpy(d_variables, variables, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factors, factors, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToFactors, variableToFactors[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    pflip<<<(nQubits+255)/256,256>>>(nQubits, d_states, d_variables, d_factors, d_variableToFactors, d_variableDegrees, maxVariableDegree);
    hipMemcpy(variables, d_variables, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(factors, d_factors, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_variables);
    hipFree(d_factors);
    hipFree(d_variableToFactors);
    hipFree(d_variableDegrees);
}

void initVariableMessagesWrap(int M, int nChecks, double* variableMessages, int* factorDegrees, int maxFactorDegree, double llrp0, double llrq0)
{
    double *d_variableMessages;
    int *d_factorDegrees;
    hipMalloc(&d_variableMessages, maxFactorDegree*M*sizeof(double));
    hipMalloc(&d_factorDegrees, M*sizeof(int));
    hipMemcpy(d_variableMessages, variableMessages, maxFactorDegree*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factorDegrees, factorDegrees, M*sizeof(double), hipMemcpyHostToDevice);
    initVariableMessages<<<(M+255)/256,256>>>(M, nChecks, d_variableMessages, d_factorDegrees, maxFactorDegree, llrp0, llrq0);
    hipMemcpy(variableMessages, d_variableMessages, maxFactorDegree*M*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_variableMessages);
    hipFree(d_factorDegrees);
}

void updateFactorMessagesTanhWrap(int N, int M, double* variableMessages, double* factorMessages, int* factors, 
        int** factorToVariables, int* factorDegrees, int maxFactorDegree, int** factorToPos, int maxVariableDegree)
{
    double *d_variableMessages, *d_factorMessages;
    int *d_factors, *d_factorToVariables, *d_factorDegrees, *d_factorToPos;
    hipMalloc(&d_variableMessages, maxFactorDegree*M*sizeof(double));
    hipMalloc(&d_factorMessages, maxVariableDegree*N*sizeof(double));
    hipMalloc(&d_factors, M*sizeof(int));
    hipMalloc(&d_factorToVariables, maxFactorDegree*M*sizeof(int));
    hipMalloc(&d_factorDegrees, M*sizeof(int));
    hipMalloc(&d_factorToPos, maxFactorDegree*M*sizeof(int));
    hipMemcpy(d_variableMessages, variableMessages, maxFactorDegree*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factorMessages, factorMessages, maxVariableDegree*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factors, factors, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToVariables, factorToVariables[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorDegrees, factorDegrees, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToPos, factorToPos[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    updateFactorMessagesTanh<<<(M+255)/256,256>>>(M, d_variableMessages, d_factorMessages, d_factors, 
            d_factorToVariables, d_factorDegrees, maxFactorDegree, d_factorToPos, maxVariableDegree);
    hipMemcpy(factorMessages, d_factorMessages, maxVariableDegree*N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_variableMessages);
    hipFree(d_factorMessages);
    hipFree(d_factors);
    hipFree(d_factorToVariables);
    hipFree(d_factorDegrees);
    hipFree(d_factorToPos);
}

void updateFactorMessagesMinSum(int alpha, int N, int M, double* variableMessages, double* factorMessages, int* factors,
        int** factorToVariables, int* factorDegrees, int maxFactorDegree, int** factorToPos, int maxVariableDegree)
{
    double *d_variableMessages, *d_factorMessages;
    int *d_factors, *d_factorToVariables, *d_factorDegrees, *d_factorToPos;
    hipMalloc(&d_variableMessages, maxFactorDegree*M*sizeof(double));
    hipMalloc(&d_factorMessages, maxVariableDegree*N*sizeof(double));
    hipMalloc(&d_factors, M*sizeof(int));
    hipMalloc(&d_factorToVariables, maxFactorDegree*M*sizeof(int));
    hipMalloc(&d_factorDegrees, M*sizeof(int));
    hipMalloc(&d_factorToPos, maxFactorDegree*M*sizeof(int));
    hipMemcpy(d_variableMessages, variableMessages, maxFactorDegree*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factorMessages, factorMessages, maxVariableDegree*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factors, factors, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToVariables, factorToVariables[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorDegrees, factorDegrees, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToPos, factorToPos[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    updateFactorMessagesMinSum<<<(M+255)/256,256>>>(alpha, M, d_variableMessages, d_factorMessages, d_factors, 
            d_factorToVariables, d_factorDegrees, maxFactorDegree, d_factorToPos, maxVariableDegree);
    hipMemcpy(factorMessages, d_factorMessages, maxVariableDegree*N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_variableMessages);
    hipFree(d_factorMessages);
    hipFree(d_factors);
    hipFree(d_factorToVariables);
    hipFree(d_factorDegrees);
    hipFree(d_factorToPos);
}

void updateVariableMessagesWrap(int N, int M, int nQubits, double* factorMessages, double* variableMessages, int** variableToFactors, 
        int* variableDegrees, int maxVariableDegree, int** variableToPos, int maxFactorDegree, int llrp0, int llrq0)
{
    double *d_factorMessages, *d_variableMessages;
    int *d_variableToFactors, *d_variableDegrees, *d_variableToPos;
    hipMalloc(&d_factorMessages, maxVariableDegree*N*sizeof(double));
    hipMalloc(&d_variableMessages, maxFactorDegree*M*sizeof(double));
    hipMalloc(&d_variableToFactors, maxVariableDegree*N*sizeof(int));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMalloc(&d_variableToPos, maxVariableDegree*N*sizeof(int));
    hipMemcpy(d_factorMessages, factorMessages, maxVariableDegree*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_variableMessages, variableMessages, maxFactorDegree*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToFactors, variableToFactors[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToPos, variableToPos[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    updateVariableMessages<<<(N+255)/256,256>>>(N, nQubits, d_factorMessages, d_variableMessages, 
            d_variableToFactors, d_variableDegrees, maxVariableDegree, d_variableToPos, maxFactorDegree, llrp0, llrq0);
    hipMemcpy(variableDegrees, d_variableDegrees, maxFactorDegree*M*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_factorMessages);
    hipFree(d_variableMessages);
    hipFree(d_variableToFactors);
    hipFree(d_variableDegrees);
    hipFree(d_variableToPos);
}

void calcMarginalsWrap(int N, int nQubits, double* marginals, double* factorMessages, int* variableDegrees, int maxVariableDegree, double llrp0, double llrq0)
{
    double *d_marginals, *d_factorMessages; 
    int *d_variableDegrees;
    hipMalloc(&d_marginals, N*sizeof(double));
    hipMalloc(&d_factorMessages, maxVariableDegree*N*sizeof(double));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMemcpy(&d_factorMessages, factorMessages, maxVariableDegree*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    calcMarginals<<<(N+255)/256,256>>>(N, nQubits, d_marginals, d_factorMessages, d_variableDegrees, maxVariableDegree, llrp0, llrq0);
    hipMemcpy(marginals, d_marginals, N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_marginals);
    hipFree(d_factorMessages);
}

void bpCorrectionWrap(int N, int M, int nQubits, int nChecks, double* marginals, 
        int* variables, int* factors, int** variableToFactors, int* variableDegrees, int maxVariableDegree)
{
    double *d_marginals;
    int *d_variables, *d_factors, *d_variableToFactors, *d_variableDegrees;
    hipMalloc(&d_marginals, N*sizeof(double));
    hipMalloc(&d_variables, N*sizeof(int));
    hipMalloc(&d_factors, M*sizeof(int));
    hipMalloc(&d_variableToFactors, maxVariableDegree*N*sizeof(int));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMemcpy(d_marginals, marginals, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_variables, variables, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factors, factors, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToFactors, variableToFactors[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    bpCorrection<<<(N+255)/256,256>>>(nQubits, nChecks, d_marginals, d_variables, d_factors, 
            d_variableToFactors, d_variableDegrees, maxVariableDegree);
    hipMemcpy(variables, d_variables, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(factors, d_factors, M*sizeof(int), hipMemcpyDeviceToHost);
}
