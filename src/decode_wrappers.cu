#include "decode.cuh"
#include "decode_wrappers.cuh"

//Only used for testing of functions as nvcc doesn't like compiling gtest

void wipeArrayWrap(int N, int* array)
{
    int *d_array;
    hipMalloc(&d_array, N*sizeof(int));
    hipMemcpy(d_array, array, N*sizeof(int), hipMemcpyHostToDevice);
    wipeArray<<<(N+255)/256,256>>>(N, d_array);
    hipMemcpy(array, d_array, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
}

void arrayErrorsWrap(int N, unsigned int seed, int* errorTarget, float errorProb)
{
    int *d_errorTarget;
    hipMalloc(&d_errorTarget, N*sizeof(int));
    hipMemcpy(d_errorTarget, errorTarget, N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    arrayErrors<<<(N+255)/256,256>>>(N, d_states, d_errorTarget, errorProb);
    hipMemcpy(errorTarget, d_errorTarget, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_errorTarget);
}

void depolErrorsWrap(int N, unsigned int seed, int* qubitsX, int* qubitsZ, float errorProb)
{
    int *d_qubitsX, *d_qubitsZ;
    hipMalloc(&d_qubitsX, N*sizeof(int));
    hipMemcpy(d_qubitsX, qubitsX, N*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&d_qubitsZ, N*sizeof(int));
    hipMemcpy(d_qubitsZ, qubitsZ, N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    depolErrors<<<(N+255)/256,256>>>(N, d_states, d_qubitsX, d_qubitsZ, errorProb);
    hipMemcpy(qubitsX, d_qubitsX, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(qubitsZ, d_qubitsZ, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qubitsX);
    hipFree(d_qubitsZ);
}

void flipWrap(int N, int M, int* qubits, int* syndrome, int** bitToChecks, int maxBitDegree)
{
    int *d_qubits, *d_syndrome, *d_bitToChecks;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_bitToChecks, maxBitDegree*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bitToChecks, bitToChecks[0], maxBitDegree*N*sizeof(int), hipMemcpyHostToDevice);
    flip<<<(N+255)/256,256>>>(N, M, d_qubits, d_syndrome, d_bitToChecks, maxBitDegree);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_bitToChecks);
}

void pflipWrap(int N, int M, unsigned int seed, int* qubits, int* syndrome, int** bitToChecks, int maxBitDegree)
{
    int *d_qubits, *d_syndrome, *d_bitToChecks;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_bitToChecks, maxBitDegree*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bitToChecks, bitToChecks[0], maxBitDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    pflip<<<(N+255)/256,256>>>(N, M, d_states, d_qubits, d_syndrome, d_bitToChecks, maxBitDegree);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_bitToChecks);
}

void calculateSyndromeWrap(int N, int M, int* qubits, int* syndrome, int** checkToBits, int maxCheckDegree)
{
    int *d_qubits, *d_syndrome, *d_checkToBits;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_checkToBits, maxCheckDegree*M*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_checkToBits, checkToBits[0], maxCheckDegree*M*sizeof(int), hipMemcpyHostToDevice);
    calculateSyndrome<<<(N+255)/256,256>>>(M, d_qubits, d_syndrome, d_checkToBits, maxCheckDegree);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_checkToBits);
}
