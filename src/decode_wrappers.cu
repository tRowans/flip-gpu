#include "decode.cuh"
#include "decode_wrappers.cuh"
#include<iostream>

//Only used for testing of functions as nvcc doesn't like compiling gtest

void wipeArrayWrap(int N, int* array)
{
    int *d_array;
    hipMalloc(&d_array, N*sizeof(int));
    hipMemcpy(d_array, array, N*sizeof(int), hipMemcpyHostToDevice);
    wipeArray<<<(N+255)/256,256>>>(N, d_array);
    hipMemcpy(array, d_array, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
}

void applyErrorsWrap(int N, unsigned int seed, int* lookup, int* errorTarget, double errorProb)
{
    int *d_lookup, *d_errorTarget;
    hipMalloc(&d_lookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_errorTarget, N*sizeof(int));
    hipMemcpy(d_lookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_errorTarget, errorTarget, N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    applyErrors<<<(N+255)/256,256>>>(d_lookup, d_states, d_errorTarget, errorProb);
    hipMemcpy(errorTarget, d_errorTarget, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_lookup);
    hipFree(d_errorTarget);
}

void calculateSyndromeWrap(int N, int* lookup, int* qubits, int* syndrome, int** edgeToFaces)
{
    int *d_lookup, *d_qubits, *d_syndrome, *d_edgeToFaces;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_lookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    calculateSyndrome<<<(N+255)/256,256>>>(d_lookup, d_qubits, d_syndrome, d_edgeToFaces);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_lookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_edgeToFaces);
}

void flipWrap(int N, int* qLookup, int* sLookup, int* qubits, int* syndrome, int** faceToEdges)
{
    int *d_qLookup, *d_sLookup, *d_qubits, *d_syndrome, *d_faceToEdges, *d_edgeToFaces;
    double *d_variableMessages, *d_qubitMarginals;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    flip<<<(N+255)/256,256>>>(d_qLookup, d_sLookup, d_qubits, d_syndrome, d_faceToEdges);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_faceToEdges);
    hipFree(d_edgeToFaces);
}

void pflipWrap(int N, unsigned int seed, int* qLookup, int* sLookup, int* qubits, int* syndrome, int** faceToEdges)
{
    int *d_qLookup, *d_sLookup, *d_qubits, *d_syndrome, *d_faceToEdges, *d_edgeToFaces;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    pflip<<<(N+255)/256,256>>>(d_qLookup, d_sLookup, d_qubits, d_syndrome, d_faceToEdges, d_states);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_faceToEdges);
    hipFree(d_edgeToFaces);
}

void initVariableMessagesWrap(int N, int* lookup, double* variableMessages, double llr0, double llrq0)
{
    int *d_sLookup;
    double *d_variableMessages;
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_variableMessages, 8*N*sizeof(double));
    hipMemcpy(d_sLookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    initVariableMessages<<<(N+255)/256,256>>>(d_sLookup, d_variableMessages, llr0, llrq0);
    hipMemcpy(variableMessages, d_variableMessages, 5*N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_sLookup);
    hipFree(d_variableMessages);
}

void updateFactorMessagesWrap(int N, int* lookup, double* variableMessages, int* syndrome, double* factorMessages, int** edgeToFaces, int** faceToEdges)
{
    int *d_sLookup, *d_syndrome, *d_edgeToFaces, *d_faceToEdges;
    double *d_variableMessages, *d_factorMessages;
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_variableMessages, 8*N*sizeof(double));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_factorMessages, 5*N*sizeof(double));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMemcpy(d_sLookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableMessages, variableMessages, 5*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    updateFactorMessages<<<(N+255)/256,256>>>(d_sLookup, d_variableMessages, d_syndrome, d_factorMessages, d_edgeToFaces, d_faceToEdges, N);
    hipMemcpy(factorMessages, d_factorMessages, 5*N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_sLookup);
    hipFree(d_variableMessages);
    hipFree(d_syndrome);
    hipFree(d_factorMessages);
    hipFree(d_edgeToFaces);
    hipFree(d_faceToEdges);
}

void updateVariableMessagesWrap(int N, int* lookup, double* variableMessages, double* factorMessages, int** faceToEdges, int** edgeToFaces, double llr0)
{
    int *d_qLookup, *d_faceToEdges, *d_edgeToFaces;
    double *d_variableMessages, *d_factorMessages;
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_variableMessages, 5*N*sizeof(double));
    hipMalloc(&d_factorMessages, 5*N*sizeof(double));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMemcpy(d_qLookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorMessages, factorMessages, 5*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    updateVariableMessages<<<(N+255)/256,256>>>(d_qLookup, d_variableMessages, d_factorMessages, d_faceToEdges, d_edgeToFaces, llr0);
    hipMemcpy(variableMessages, d_variableMessages, 5*N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_variableMessages);
    hipFree(d_factorMessages);
    hipFree(d_faceToEdges);
    hipFree(d_edgeToFaces);
}

void calcMarginalsWrap(int N, int* qLookup, int* sLookup, double* qubitMarginals, double* stabMarginals, double* factorMessages, double llr0, double llrq0)
{
    int *d_qLookup, *d_sLookup;
    double *d_qubitMarginals, *d_stabMarginals, *d_factorMessages;
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_qubitMarginals, N*sizeof(double));
    hipMalloc(&d_stabMarginals, N*sizeof(double));
    hipMalloc(&d_factorMessages, 5*N*sizeof(double));
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qubitMarginals, qubitMarginals, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_stabMarginals, stabMarginals, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factorMessages, factorMessages, 5*N*sizeof(double), hipMemcpyHostToDevice);
    calcMarginals<<<(N+255)/256,256>>>(d_qLookup, d_sLookup, d_qubitMarginals, d_stabMarginals, d_factorMessages, llr0, llrq0, N);
    hipMemcpy(qubitMarginals, d_qubitMarginals, N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(stabMarginals, d_stabMarginals, N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubitMarginals);
    hipFree(d_stabMarginals);
    hipFree(d_factorMessages);
}

void bpCorrectionWrap(int N, int* qLookup, int* sLookup, int* qubits, double* qubitMarginals, int* syndrome, double* stabMarginals, int* faceToEdges);
{
    int *d_qLookup, *d_sLookup, *d_qubits, *d_syndrome, *d_faceToEdges;
    double *d_qubitMarginals, *d_stabMarginals;
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMalloc(&d_qubitMarginals, N*sizeof(double));
    hipMalloc(&d_stabMarginals, N*sizeof(double));
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qubitMarginals, qubitMarginals, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_stabMarginals, stabMarginals, N*sizeof(double), hipMemcpyHostToDevice);
    bpCorrection<<<(N+255)/256,256>>>(d_qLookup, d_sLookup, d_qubits, d_qubitMarginals, d_syndrome, d_stabMarginals, d_faceToEdges);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubits);
    hipFree(d_qubitMarginals);
    hipFree(d_syndrome);
    hipFree(d_stabMarginals);
    hipFree(d_faceToEdges);
}

void measureLogicalsWrap(int N, int* lookup, int* qubits, int &nOdd, int L, char bounds)
{
    int *d_lookup, *d_qubits, *d_nOdd;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_lookup, ((3*L*L+63)/64)*64*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lookup, lookup, ((3*L*L+63)/64)*64*sizeof(int), hipMemcpyHostToDevice);
    hipMallocManaged(&d_nOdd, sizeof(int));
    *d_nOdd = 0;
    measureLogicals<<<(3*L*L+63)/64,64>>>(d_lookup, d_qubits, d_nOdd, L, bounds);
    hipDeviceSynchronize();
    nOdd = *d_nOdd;
    hipFree(d_lookup);
    hipFree(d_qubits);
    hipFree(d_nOdd);
}
