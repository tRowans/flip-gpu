#include "decode.cuh"
#include "decode_wrappers.cuh"

//Only used for testing of functions as nvcc doesn't like compiling gtest

void wipeArrayWrap(int maxIndex, int* array)
{
    int *d_array;
    hipMalloc(&d_array, maxIndex*sizeof(int));
    hipMemcpy(d_array, array, maxIndex*sizeof(int), hipMemcpyHostToDevice);
    wipeArray<<<(N+255)/256,256>>>(maxIndex, d_array);
    hipMemcpy(array, d_array, maxIndex*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
}

void arrayErrorsWrap(int maxIndex, unsigned int seed, int* errorTarget, float errorProb)
{
    int *d_errorTarget;
    hipMalloc(&d_errorTarget, maxIndex*sizeof(int));
    hipMemcpy(d_errorTarget, errorTarget, maxIndex*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, maxIndex*sizeof(hiprandState_t));
    createStates<<<(maxIndex+255)/256,256>>>(maxIndex, seed, d_states);
    arrayErrors<<<(maxIndex+255)/256,256>>>(maxIndex, d_states, d_errorTarget, errorProb);
    hipMemcpy(errorTarget, d_errorTarget, maxIndex*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_errorTarget);
}

void depolErrorsWrap(int nQubits, unsigned int seed, int* qubitsX, int* qubitsZ, float errorProb)
{
    int *d_qubitsX, *d_qubitsZ;
    hipMalloc(&d_qubitsX, nQubits*sizeof(int));
    hipMemcpy(d_qubitsX, qubitsX, nQubits*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&d_qubitsZ, nQubits*sizeof(int));
    hipMemcpy(d_qubitsZ, qubitsZ, nQubits*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, nQubits*sizeof(hiprandState_t));
    createStates<<<(nQubits+255)/256,256>>>(nQubits, seed, d_states);
    depolErrors<<<(nQubits+255)/256,256>>>(nQubits, d_states, d_qubitsX, d_qubitsZ, errorProb);
    hipMemcpy(qubitsX, d_qubitsX, nQubits*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(qubitsZ, d_qubitsZ, nQubits*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qubitsX);
    hipFree(d_qubitsZ);
}

void calculateSyndromeWrap(int M, int nQubits, int nChecks, int* qubits, int* syndrome, int** factorToVariables, int* factorDegrees, int maxCheckDegree)
{
    int *d_qubits, *d_syndrome, *d_factorToVariables, *d_factorDegrees;
    hipMalloc(&d_qubits, nQubits*sizeof(int));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_factorToVariables, maxFactorDegree*M*sizeof(int));
    hipMalloc(&d_factorDegrees, M*sizeof(int));
    hipMemcpy(d_qubits, qubits, nQubits*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToVariables, factorToVariables[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorDegrees, factorDegrees, M*sizeof(int), hipMemcpyHostToDevice);
    calculateSyndrome<<<(nChecks+255)/256,256>>>(nChecks, d_qubits, d_syndrome, d_factorToVariables, d_factorDegrees, maxFactorDegree);
    hipMemcpy(qubits, d_qubits, nQubits*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_factorToVariables);
    hipFree(d_factorDegrees);
}

void flipWrap(int M, int N, int nQubits, int nChecks, int* qubits, int* syndrome, int** variableToFactors, int* variableDegrees, int maxVariableDegree)
{
    int *d_qubits, *d_syndrome, *d_variableToFactors, *d_variableDegrees;
    hipMalloc(&d_qubits, nQubits*sizeof(int));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_variableToFactors, maxVariableDegree*N*sizeof(int));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMemcpy(d_qubits, qubits, nQubits*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToFactors, variableToFactors[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    flip<<<(nQubits+255)/256,256>>>(nQubits, d_qubits, d_syndrome, d_variableToFactors, d_variableDegrees, maxVariableDegree);
    hipMemcpy(qubits, d_qubits, nQubits*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_variableToFactors);
    hipFree(d_variableDegrees);
}

void pflipWrap(int M, int N, int nQubits, int nChecks, unsigned int seed, int* qubits, int* syndrome, 
        int** variableToFactors, int* variableDegrees, int maxVariableDegree)
{
    int *d_qubits, *d_syndrome, *d_variableToFactors, *d_variableDegrees;
    hipMalloc(&d_qubits, nQubits*sizeof(int));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_variableToFactors, maxVariableDegree*N*sizeof(int));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMemcpy(d_qubits, qubits, nQubits*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToFactors, variableToFactors[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, nQubits*sizeof(hiprandState_t));
    createStates<<<(nQubits+255)/256,256>>>(nQubits, seed, d_states);
    pflip<<<(nQubits+255)/256,256>>>(nQubits, d_states, d_qubits, d_syndrome, d_variableToFactors, d_variableDegrees, maxVariableDegree);
    hipMemcpy(qubits, d_qubits, nQubits*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, M*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_variableToFactors);
    hipFree(d_variableDegrees);
}

void initVariableMessagesWrap(int M, int nChecks, double** variableMessages, int* factorDegrees, int maxFactorDegree, double llrp0, double llrq0)
{
    double *d_variableMessages;
    int *d_factorDegrees;
    hipMalloc(&d_variableMessages, maxFactorDegree*M*sizeof(double));
    hipMalloc(&d_factorDegrees, M*sizeof(int));
    hipMemcpy(d_variableMessages, variableMessages[0], maxFactorDegree*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factorDegrees, factorDegrees, M*sizeof(double), hipMemcpyHostToDevice);
    initVariableMessages<<<(M+255)/256,256>>>(M, nChecks, d_variableMessages, d_factorDegrees, maxFactorDegree, llrp0, llrq0);
    hipMemcpy(variableMessages[0], d_variableMessages, maxFactorDegree*M*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_variableMessages);
    hipFree(d_factorDegrees);
}

void updateFactorMessagesTanhWrap(int M, int N, double** variableMessages, double** factorMessages, int* syndrome, 
        int** factorToVariables, int* factorDegrees, int maxFactorDegree, int** factorToPos, int maxVariableDegree)
{
    double *d_variableMessages, *d_factorMessages;
    int *d_syndrome, *d_factorToVariables, *d_factorDegrees, *d_factorToPos;
    hipMalloc(&d_variableMessages, maxFactorDegree*M*sizeof(double));
    hipMalloc(&d_factorMessages, maxVariableDegree*N*sizeof(double));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_factorToVariables, maxFactorDegree*M*sizeof(int));
    hipMalloc(&d_factorDegrees, M*sizeof(int));
    hipMalloc(&d_factorToPos, maxFactorDegree*M*sizeof(int));
    hipMemcpy(d_variableMessages, variableMessages[0], maxFactorDegree*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factorMessages, factorMessages[0], maxVariableDegree*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToVariables, factorToVariables[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorDegrees, factorDegrees, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToPos, factorToPos[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    updateFactorMessagesTanh<<<(M+255)/256,256>>>(M, d_variableMessages, d_factorMessages, d_syndrome, 
            d_factorToVariables, d_factorDegrees, maxFactorDegree, d_factorToPos, maxVariableDegree);
    hipMemcpy(factorMessages[0], d_factorMessages, maxVariableDegree*N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_variableMessages);
    hipFree(d_factorMessages);
    hipFree(d_syndrome);
    hipFree(d_factorToVariables);
    hipFree(d_factorDegrees);
    hipFree(d_factorToPos);
}

void updateFactorMessagesMinSum(int alpha, int M, int N, double** variableMessages, double** factorMessages, int* syndrome
        int** factorToVariables, int* factorDegrees, int maxFactorDegree, int** factorToPos, int maxVariableDegree)
{
    double *d_variableMessages, *d_factorMessages;
    int *d_syndrome, *d_factorToVariables, *d_factorDegrees, *d_factorToPos;
    hipMalloc(&d_variableMessages, maxFactorDegree*M*sizeof(double));
    hipMalloc(&d_factorMessages, maxVariableDegree*N*sizeof(double));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_factorToVariables, maxFactorDegree*M*sizeof(int));
    hipMalloc(&d_factorDegrees, M*sizeof(int));
    hipMalloc(&d_factorToPos, maxFactorDegree*M*sizeof(int));
    hipMemcpy(d_variableMessages, variableMessages[0], maxFactorDegree*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_factorMessages, factorMessages[0], maxVariableDegree*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToVariables, factorToVariables[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorDegrees, factorDegrees, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_factorToPos, factorToPos[0], maxFactorDegree*M*sizeof(int), hipMemcpyHostToDevice);
    updateFactorMessagesMinSum<<<(M+255)/256,256>>>(alpha, M, d_variableMessages, d_factorMessages, d_syndrome, 
            d_factorToVariables, d_factorDegrees, maxFactorDegree, d_factorToPos, maxVariableDegree);
    hipMemcpy(factorMessages[0], d_factorMessages, maxVariableDegree*N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_variableMessages);
    hipFree(d_factorMessages);
    hipFree(d_syndrome);
    hipFree(d_factorToVariables);
    hipFree(d_factorDegrees);
    hipFree(d_factorToPos);
}

void updateVariableMessagesWrap(int M, int N, int nQubits, double** factorMessages, double** variableMessages, int** variableToFactors, 
        int* variableDegrees, int maxVariableDegree, int** variableToPos, int maxFactorDegree, int llrp0, int llrq0)
{
    double *d_factorMessages, *d_variableMessages;
    int *d_variableToFactors, *d_variableDegrees, *d_variableToPos;
    hipMalloc(&d_factorMessages, maxVariableDegree*N*sizeof(double));
    hipMalloc(&d_variableMessages, maxFactorDegree*M*sizeof(double));
    hipMalloc(&d_variableToFactors, maxVariableDegree*N*sizeof(int));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMalloc(&d_variableToPos, maxVariableDegree*N*sizeof(int));
    hipMemcpy(d_factorMessages, factorMessages[0], maxVariableDegree*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_variableMessages, variableMessages[0], maxFactorDegree*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToFactors, variableToFactors[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToPos, variableToPos[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    updateVariableMessages<<<(N+255)/256,256>>>(N, nQubits, d_factorMessages, d_variableMessages, 
            d_variableToFactors, d_variableDegrees, maxVariableDegree, d_variableToPos, maxFactorDegree, llrp0, llrq0);
    hipMemcpy(variableDegrees[0], d_variableDegrees, maxFactorDegree*M*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_factorMessages);
    hipFree(d_variableMessages);
    hipFree(d_variableToFactors);
    hipFree(d_variableDegrees);
    hipFree(d_variableToPos);
}

void calcMarginalsWrap(int N, int nQubits, double* marginals, double** factorMessages, double llrp0, double llrq0, int maxVariableDegree)
{
    double d_marginals, d_factorMessages;
    hipMalloc(&d_marginals, N*sizeof(double));
    hipMalloc(&d_factorMessages, maxVariableDegree*N*sizeof(double));
    hipMemcpy(&d_factorMessages, factorMessages[0], maxVariableDegree*N*sizeof(double), hipMemcpyHostToDevice);
    calcMarginals<<<(N+255)/256,256>>>(N, nQubits, d_marginals, d_factorMessages, llrp0, llrq0);
    hipMemcpy(marginals, d_marginals, N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_marginals);
    hipFree(d_factorMessages);
}

void bpCorrectionWrap(int M, int N, int nQubits, int nChecks, double* marginals, 
        int* qubits, int* syndrome, int** variableToFactors, int* variableDegrees, int maxVariableDegree)
{
    double *d_marginals;
    int *d_qubits, *d_syndrome, *d_variableToFactors, *d_variableDegrees;
    hipMalloc(&d_marginals, N*sizeof(double));
    hipMalloc(&d_qubits, nQubits*sizeof(int));
    hipMalloc(&d_syndrome, M*sizeof(int));
    hipMalloc(&d_variableToFactors, maxVariableDegree*N*sizeof(int));
    hipMalloc(&d_variableDegrees, N*sizeof(int));
    hipMemcpy(d_marginals, marginals, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_qubits, qubits, nQubits*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableToFactors, variableToFactors[0], maxVariableDegree*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_variableDegrees, variableDegrees, N*sizeof(int), hipMemcpyHostToDevice);
    bpCorrection<<<(N+255)/256,256>>>(N, nQubits, nChecks, d_marginals, d_qubits, d_syndrome, 
            d_variableToFactors, d_variableDegrees, maxVariableDegree);
    hipMemcpy(qubits, d_qubits, nQubits*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, M*sizeof(int), hipMemcpyDeviceToHost);
}
