#include "decode.cuh"
#include "decode_wrappers.cuh"

//Only used for testing of functions as nvcc doesn't like compiling gtest

void wipeArrayWrap(int N, int* array)
{
    int *d_array;
    hipMalloc(&d_array, N*sizeof(int));
    hipMemcpy(d_array, array, N*sizeof(int), hipMemcpyHostToDevice);
    wipeArray<<<(N+255)/256,256>>>(N, d_array);
    hipMemcpy(array, d_array, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);
}

void applyErrorsWrap(int N, unsigned int seed, int* lookup, int* errorTarget, float errorProb)
{
    int *d_lookup, *d_errorTarget;
    hipMalloc(&d_lookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_errorTarget, N*sizeof(int));
    hipMemcpy(d_lookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_errorTarget, errorTarget, N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    applyErrors<<<(N+255)/256,256>>>(d_lookup, d_states, d_errorTarget, errorProb);
    hipMemcpy(errorTarget, d_errorTarget, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_lookup);
    hipFree(d_errorTarget);
}

void flipWrap(int N, int* qLookup, int* sLookup, int* qubits, int* syndrome, int** faceToEdges)
{
    int *d_qLookup, *d_sLookup, *d_qubits, *d_syndrome, *d_faceToEdges;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    flip<<<(N+255)/256,256>>>(d_qLookup, d_sLookup, d_qubits, d_syndrome, d_faceToEdges);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_faceToEdges);
}

void pflipWrap(int N, unsigned int seed, int* qLookup, int* sLookup, int* qubits, int* syndrome, int** faceToEdges)
{
    int *d_qLookup, *d_sLookup, *d_qubits, *d_syndrome, *d_faceToEdges;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t *d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    pflip<<<(N+255)/256,256>>>(d_qLookup, d_sLookup, d_qubits, d_syndrome, d_faceToEdges, d_states);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_faceToEdges);
}

void edgeFlipWrap(int N, unsigned int seed, int* qLookup, int* sLookup, 
                      int* qubits, int* syndrome, int** edgeToFaces, int** faceToEdges)
{
    int *d_qLookup, *d_sLookup, *d_qubits, *d_syndrome, *d_edgeToFaces, *d_faceToEdges;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_qLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_sLookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMalloc(&d_faceToEdges, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_qLookup, qLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sLookup, sLookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces, 4*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_faceToEdges, faceToEdges, 4*N*sizeof(int), hipMemcpyHostToDevice);
    hiprandState_t * d_states;
    hipMalloc(&d_states, N*sizeof(hiprandState_t));
    createStates<<<(N+255)/256,256>>>(N, seed, d_states);
    edgeFlip<<<(N+255)/256,256>>>(d_qLookup,d_sLookup, d_qubits, d_syndrome, d_edgeToFaces, d_faceToEdges, d_states);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_qLookup);
    hipFree(d_sLookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_edgeToFaces);
    hipFree(d_faceToEdges);
}

void calculateSyndromeWrap(int N, int* lookup, int* qubits, int* syndrome, int** edgeToFaces)
{
    int *d_lookup, *d_qubits, *d_syndrome, *d_edgeToFaces;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_syndrome, N*sizeof(int));
    hipMalloc(&d_lookup, ((N+255)/256)*256*sizeof(int));
    hipMalloc(&d_edgeToFaces, 4*N*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_syndrome, syndrome, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lookup, lookup, ((N+255)/256)*256*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeToFaces, edgeToFaces[0], 4*N*sizeof(int), hipMemcpyHostToDevice);
    calculateSyndrome<<<(N+255)/256,256>>>(d_lookup, d_qubits, d_syndrome, d_edgeToFaces);
    hipMemcpy(qubits, d_qubits, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(syndrome, d_syndrome, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_lookup);
    hipFree(d_qubits);
    hipFree(d_syndrome);
    hipFree(d_edgeToFaces);
}

void measureLogicalsWrap(int N, int* lookup, int* qubits, int &nOdd, int L, char bounds)
{
    int *d_lookup, *d_qubits, *d_nOdd;
    hipMalloc(&d_qubits, N*sizeof(int));
    hipMalloc(&d_lookup, ((3*L*L+63)/64)*64*sizeof(int));
    hipMemcpy(d_qubits, qubits, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lookup, lookup, ((3*L*L+63)/64)*64*sizeof(int), hipMemcpyHostToDevice);
    hipMallocManaged(&d_nOdd, sizeof(int));
    *d_nOdd = 0;
    measureLogicals<<<(3*L*L+63)/64,64>>>(d_lookup, d_qubits, d_nOdd, L, bounds);
    hipDeviceSynchronize();
    nOdd = *d_nOdd;
    hipFree(d_lookup);
    hipFree(d_qubits);
    hipFree(d_nOdd);
}
